/*
 * Overdamped Brownian particle in symmetric piecewise linear potential
 *
 * \dot{x} = -V'(x) + Gaussian, Poissonian and dichotomous noise
 *
 */

#include <stdio.h>
#include <getopt.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define PI 3.14159265358979f

//model
__constant__ float d_Dg, d_Dp, d_lambda, d_mean, d_fa, d_fb, d_mua, d_mub;
__constant__ int d_comp;
float h_lambda, h_fa, h_fb, h_mua, h_mub, h_mean;
int h_comp;

//simulation
float h_trans;
int h_dev, h_block, h_grid, h_spp, h_samples;
long h_paths, h_periods, h_threads, h_steps, h_trigger;
__constant__ int d_spp, d_samples;
__constant__ long d_paths;

//output
char *h_domain;
char h_domainx;
float h_beginx, h_endx;
int h_logx, h_points, h_moments;
__constant__ char d_domainx;
__constant__ int d_points;

//vector
float *h_x, *h_xb, *h_fx, *h_dx;
float *d_x, *d_fx, *d_dx;
int *d_pcd, *d_dcd, *d_dst;
unsigned int *h_seeds, *d_seeds;
hiprandState *d_states;

size_t size_f, size_i, size_ui, size_p;
hiprandGenerator_t gen;

static struct option options[] = {
    {"Dg", required_argument, NULL, 'a'},
    {"Dp", required_argument, NULL, 'b'},
    {"lambda", required_argument, NULL, 'c'},
    {"fa", required_argument, NULL, 'd'},
    {"fb", required_argument, NULL, 'e'},
    {"mua", required_argument, NULL, 'f'},
    {"mub", required_argument, NULL, 'g'},
    {"comp", required_argument, NULL, 'h'},
    {"mean", required_argument, NULL, 'i'},
    {"dev", required_argument, NULL, 'j'},
    {"block", required_argument, NULL, 'k'},
    {"paths", required_argument, NULL, 'l'},
    {"periods", required_argument, NULL, 'm'},
    {"trans", required_argument, NULL, 'n'},
    {"spp", required_argument, NULL, 'o'},
    {"samples", required_argument, NULL, 'p'},
    {"mode", required_argument, NULL, 'q'},
    {"domain", required_argument, NULL, 'r'},
    {"domainx", required_argument, NULL, 's'},
    {"logx", required_argument, NULL, 't'},
    {"points", required_argument, NULL, 'u'},
    {"beginx", required_argument, NULL, 'v'},
    {"endx", required_argument, NULL, 'w'}
};

void usage(char **argv)
{
    printf("Usage: %s <params> \n\n", argv[0]);
    printf("Model params:\n");
    printf("    -a, --Dg=FLOAT          set the Gaussian noise intensity 'D_G' to FLOAT\n");
    printf("    -b, --Dp=FLOAT          set the Poissonian noise intensity 'D_P' to FLOAT\n");
    printf("    -c, --lambda=FLOAT      set the Poissonian kicks frequency '\\lambda' to FLOAT\n");
    printf("    -d, --fa=FLOAT          set the first state of the dichotomous noise 'F_a' to FLOAT\n");
    printf("    -e, --fb=FLOAT          set the second state of the dichotomous noise 'F_b' to FLOAT\n");
    printf("    -f, --mua=FLOAT         set the transition rate of the first state of dichotomous noise '\\mu_a' to FLOAT\n");
    printf("    -g, --mub=FLOAT         set the transition rate of the second state of dichotomous noise '\\mu_b' to FLOAT\n");
    printf("    -h, --comp=INT          choose between biased and unbiased Poissonian or dichotomous noise. INT can be one of:\n");
    printf("                            0: biased; 1: unbiased\n");
    printf("    -i, --mean=FLOAT        if is nonzero, fix the mean value of Poissonian noise or dichotomous noise to FLOAT, matters only for domains p, l, a, b, m or n\n");
    printf("Simulation params:\n");
    printf("    -j, --dev=INT           set the gpu device to INT\n");
    printf("    -k, --block=INT         set the gpu block size to INT\n");
    printf("    -l, --paths=LONG        set the number of paths to LONG\n");
    printf("    -m, --periods=LONG      set the number of periods to LONG\n");
    printf("    -n, --trans=FLOAT       specify fraction FLOAT of periods which stands for transients\n");
    printf("    -o, --spp=INT           specify how many integration steps should be calculated for the smallest characteristic time scale\n");
    printf("    -p, --samples=INT       specify how many integration steps should be calculated for a single kernel call\n");
    printf("Output params:\n");
    printf("    -q, --mode=STRING       sets the output mode. STRING can be one of:\n");
    printf("                            moments: the first moment <<v>>\n");
    printf("    -r, --domain=STRING     simultaneously scan over one or two model params. STRING can be one of:\n");
    printf("                            1d: only one parameter\n");
    printf("    -s, --domainx=CHAR      sets the first domain of the moments. CHAR can be one of:\n");
    printf("                            D: Dg; p: Dp; l: lambda; a: fa; b: fb; m: mua; n: mub\n");
    printf("    -t, --logx=INT          choose between linear and logarithmic scale of the domainx\n");
    printf("                            0: linear; 1: logarithmic\n");
    printf("    -u, --points=INT        set the number of samples to generate between begin and end\n");
    printf("    -v, --beginx=FLOAT      set the starting value of the domainx to FLOAT\n");
    printf("    -w, --endx=FLOAT        set the end value of the domainx to FLOAT\n");
    printf("\n");
}

void parse_cla(int argc, char **argv)
{
    float ftmp;
    int c, itmp;

    while( (c = getopt_long(argc, argv, "a:b:c:d:e:f:g:h:i:j:k:l:m:n:o:p:q:r:s:t:u:v:w", options, NULL)) != EOF) {
        switch (c) {
            case 'a':
                ftmp = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_Dg), &ftmp, sizeof(float));
                break;
            case 'b':
                ftmp = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_Dp), &ftmp, sizeof(float));
                break;
            case 'c':
                h_lambda = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_lambda), &h_lambda, sizeof(float));
                break;
            case 'd':
                h_fa = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_fa), &h_fa, sizeof(float));
                break;
            case 'e':
                h_fb = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_fb), &h_fb, sizeof(float));
                break;
            case 'f':
                h_mua = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_mua), &h_mua, sizeof(float));
                break;
            case 'g':
                h_mub = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_mub), &h_mub, sizeof(float));
                break;
            case 'h':
                h_comp = atoi(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_comp), &h_comp, sizeof(int));
                break;
            case 'i':
                h_mean = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_mean), &h_mean, sizeof(float));
                break;
            case 'j':
                itmp = atoi(optarg);
                hipSetDevice(itmp);
                break;
            case 'k':
                h_block = atoi(optarg);
                break;
            case 'l':
                h_paths = atol(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_paths), &h_paths, sizeof(long));
                break;
            case 'm':
                h_periods = atol(optarg);
                break;
            case 'n':
                h_trans = atof(optarg);
                break;
            case 'o':
                h_spp = atoi(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_spp), &h_spp, sizeof(int));
                break;
            case 'p':
                h_samples = atoi(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_samples), &h_samples, sizeof(int));
                break;
            case 'q':
                if ( !strcmp(optarg, "moments") ) {
                    h_moments = 1;
                }
                break;
            case 'r':
                h_domain = optarg;
                break;
            case 's':
                h_domainx = optarg[0]; 
                hipMemcpyToSymbol(HIP_SYMBOL(d_domainx), &h_domainx, sizeof(char));
                break;
            case 't':
                h_logx = atoi(optarg);
                break;
            case 'u':
                h_points = atoi(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_points), &h_points, sizeof(int));
                break;
            case 'v':
                h_beginx = atof(optarg);
                break;
            case 'w':
                h_endx = atof(optarg);
                break;
        }
    }
}

__global__ void init_dev_rng(unsigned int *d_seeds, hiprandState *d_states)
{
    long idx = blockIdx.x * blockDim.x + threadIdx.x;

    hiprand_init(d_seeds[idx], 0, 0, &d_states[idx]);
}

__device__ float drift(float l_x)
{    
    if (-sinf(PI*l_x) < 0.0f) {
        return -1.0f;
    } else {
        return 1.0f;
    }
}

__device__ float diffusion(float l_Dg, float l_dt, hiprandState *l_state)
{
    if (l_Dg != 0.0f) {
        float r = hiprand_uniform(l_state);
        float g = sqrtf(2.0f*l_Dg);
        if ( r <= 1.0f/6.0f ) {
            return -g*sqrtf(3.0f*l_dt);
        } else if ( r > 1.0f/6.0f && r <= 1.0f/3.0f ) {
            return g*sqrtf(3.0f*l_dt);
        } else {
            return 0.0f;
        }
    } else {
        return 0.0f;
    }
}

__global__ void init_noise(float *d_dx, int *d_pcd, int *d_dcd, int *d_dst, hiprandState *d_states)
//init noise
{
    long idx = blockIdx.x * blockDim.x + threadIdx.x;
    float l_dx; 
    hiprandState l_state;

    //cache model parameters in local variables
    l_state = d_states[idx];

    float l_Dp, l_lambda, l_mean, l_fa, l_fb, l_mua, l_mub;
    int l_comp;

    l_Dp = d_Dp;
    l_lambda = d_lambda;
    l_comp = d_comp;
    l_mean = d_mean;
    l_fa = d_fa;
    l_fb = d_fb;
    l_mua = d_mua;
    l_mub = d_mub;

    long ridx = (idx/d_paths) % d_points;
    l_dx = d_dx[ridx];

    switch(d_domainx) {
        case 'p':
            l_Dp = l_dx;
            if (l_mean != 0.0f) l_lambda = (l_mean*l_mean)/l_Dp;
            break;
        case 'l':
            l_lambda = l_dx;
            if (l_mean != 0.0f) l_Dp = (l_mean*l_mean)/l_lambda;
            break;
        case 'a':
            l_fa = l_dx;
            if (l_comp == 1) {
                l_fb = -l_fa*l_mub/l_mua;
            } else if (l_mean != 0.0f) {
                l_fb = (l_mean*(l_mua + l_mub) - l_fa*l_mub)/l_mua;
            }
            break;
        case 'b':
            l_fb = l_dx;
            if (l_comp == 1) {
                l_fa = -l_fb*l_mua/l_mub;
            } else if (l_mean != 0.0f) {
                l_fa = (l_mean*(l_mua + l_mub) - l_fb*l_mua)/l_mub;
            }
            break;
        case 'm':
            l_mua = l_dx;
            if (l_comp == 1) {
                l_mub = -l_fb*l_mua/l_fa;
            } else if (l_mean != 0.0f) {
                l_mub = (l_fb - l_mean)*l_mua/(l_mean - l_fa);
            }
            break;
        case 'n':
            l_mub = l_dx;
            if (l_comp == 1) {
                l_mua = -l_fa*l_mub/l_fb;
            } else if (l_mean != 0.0f) {
                l_mua = (l_fa - l_mean)*l_mub/(l_mean - l_fb);
            }
            break;
    }

    //step size
    float l_dt;
    int l_spp;

    l_spp = d_spp;

    if (l_lambda != 0.0f) {
        l_dt = 1.0f/l_lambda/l_spp;
    }

    if (l_mua != 0.0f) {
        float taua, taub;

        taua = 1.0f/l_mua;
        taub = 1.0f/l_mub;

        if (taua < taub) {
            l_dt = taua/l_spp;
        } else {
            l_dt = taub/l_spp;
        }
    }

    //jump countdowns
    int l_pcd, l_dcd, l_dst;
    
    if (l_lambda != 0.0f) l_pcd = (int) floorf( -logf( hiprand_uniform(&l_state) )/l_lambda/l_dt + 0.5f );

    if (l_mua != 0.0f) {
        float rn;
        rn = hiprand_uniform(&l_state);

        if (rn < 0.5f) {
            l_dst = 0;
            l_dcd = (int) floorf( -logf( hiprand_uniform(&l_state) )/l_mua/l_dt + 0.5f);
        } else {
            l_dst = 1;
            l_dcd = (int) floorf( -logf( hiprand_uniform(&l_state) )/l_mub/l_dt + 0.5f);
        }
    }
    
    //write back noise states to the global memory
    d_pcd[idx] = l_pcd;
    d_dcd[idx] = l_dcd;
    d_dst[idx] = l_dst;
    d_states[idx] = l_state;
}

__device__ float adapted_jump_poisson(int &npcd, int pcd, float l_lambda, float l_Dp, int l_comp, float l_dt, hiprandState *l_state)
{
    if (l_lambda != 0.0f) {
        if (pcd <= 0) {
            float ampmean = sqrtf(l_lambda/l_Dp);
           
            npcd = (int) floorf( -logf( hiprand_uniform(l_state) )/l_lambda/l_dt + 0.5f );

            if (l_comp) {
                float comp = sqrtf(l_Dp*l_lambda)*l_dt;
                
                return -logf( hiprand_uniform(l_state) )/ampmean - comp;
            } else {
                return -logf( hiprand_uniform(l_state) )/ampmean;
            }
        } else {
            npcd = pcd - 1;
            if (l_comp) {
                float comp = sqrtf(l_Dp*l_lambda)*l_dt;
                
                return -comp;
            } else {
                return 0.0f;
            }
        }
    } else {
        return 0.0f;
    }
}

__device__ float adapted_jump_dich(int &ndcd, int dcd, int &ndst, int dst, float l_fa, float l_fb, float l_mua, float l_mub, float l_dt, hiprandState *l_state)
{
    if (l_mua != 0.0f) {
        if (dcd <= 0) {
            if (dst == 0) {
                ndst = 1; 
                ndcd = (int) floorf( -logf( hiprand_uniform(l_state) )/l_mub/l_dt + 0.5f );
                return l_fb*l_dt;
            } else {
                ndst = 0;
                ndcd = (int) floorf( -logf( hiprand_uniform(l_state) )/l_mua/l_dt + 0.5f );
                return l_fa*l_dt;
            }
        } else {
            ndcd = dcd - 1;
            if (dst == 0) {
                return l_fa*l_dt;
            } else {
                return l_fb*l_dt;
            }
        }
    } else {
        return 0.0f;
    }
}

__device__ void predcorr(float &corrl_x, float l_x, int &npcd, int pcd, hiprandState *l_state, \
                         float l_Dg, float l_Dp, float l_lambda, int l_comp, \
                         int &ndcd, int dcd, int &ndst, int dst, float l_fa, float l_fb, float l_mua, float l_mub, float l_dt)
/* simplified weak order 2.0 adapted predictor-corrector scheme
( see E. Platen, N. Bruti-Liberati; Numerical Solution of Stochastic Differential Equations with Jumps in Finance; Springer 2010; p. 503, p. 532 )
*/
{
    float l_xt, l_xtt, predl_x;

    l_xt = drift(l_x);

    predl_x = l_x + l_xt*l_dt + diffusion(l_Dg, l_dt, l_state);

    l_xtt = drift(predl_x);

    predl_x = l_x + 0.5f*(l_xt + l_xtt)*l_dt + diffusion(l_Dg, l_dt, l_state);

    l_xtt = drift(predl_x);

    corrl_x = l_x + 0.5f*(l_xt + l_xtt)*l_dt + adapted_jump_dich(ndcd, dcd, ndst, dst, l_fa, l_fb, l_mua, l_mub, l_dt, l_state) + diffusion(l_Dg, l_dt, l_state) + adapted_jump_poisson(npcd, pcd, l_lambda, l_Dp, l_comp, l_dt, l_state);
}

__global__ void fold(float *d_x, float *d_fx)
{
    long idx = blockIdx.x * blockDim.x + threadIdx.x;
    float l_x, l_fx, f;

    l_x = d_x[idx];
    l_fx = d_fx[idx];

    f = floorf(l_x/2.0f)*2.0f;
    l_x = l_x - f;
    l_fx = l_fx + f;

    d_x[idx] = l_x;
    d_fx[idx] = l_fx;
}

void unfold(float *x, float *fx)
{
    int i;

    for (i = 0; i < h_threads; i++) {
        x[i] = x[i] + fx[i];
    }
}

__global__ void run_moments(float *d_x, float *d_dx, int *d_pcd, int *d_dcd, int *d_dst, hiprandState *d_states)
//actual moments kernel
{
    long idx = blockIdx.x * blockDim.x + threadIdx.x;
    float l_x, l_dx; 
    hiprandState l_state;

    //cache path and model parameters in local variables
    l_x = d_x[idx];
    l_state = d_states[idx];

    float l_Dg, l_Dp, l_lambda, l_mean, l_fa, l_fb, l_mua, l_mub;
    int l_comp;

    l_Dg = d_Dg;
    l_Dp = d_Dp;
    l_lambda = d_lambda;
    l_comp = d_comp;
    l_mean = d_mean;
    l_fa = d_fa;
    l_fb = d_fb;
    l_mua = d_mua;
    l_mub = d_mub;

    //run simulation for multiple values of the system parameters
    long ridx = (idx/d_paths) % d_points;
    l_dx = d_dx[ridx];

    switch(d_domainx) {
        case 'D':
            l_Dg = l_dx;
            break;
        case 'p':
            l_Dp = l_dx;
            if (l_mean != 0.0f) l_lambda = (l_mean*l_mean)/l_Dp;
            break;
        case 'l':
            l_lambda = l_dx;
            if (l_mean != 0.0f) l_Dp = (l_mean*l_mean)/l_lambda;
            break;
        case 'a':
            l_fa = l_dx;
            if (l_comp == 1) {
                l_fb = -l_fa*l_mub/l_mua;
            } else if (l_mean != 0.0f) {
                l_fb = (l_mean*(l_mua + l_mub) - l_fa*l_mub)/l_mua;
            }
            break;
        case 'b':
            l_fb = l_dx;
            if (l_comp == 1) {
                l_fa = -l_fb*l_mua/l_mub;
            } else if (l_mean != 0.0f) {
                l_fa = (l_mean*(l_mua + l_mub) - l_fb*l_mua)/l_mub;
            }
            break;
        case 'm':
            l_mua = l_dx;
            if (l_comp == 1) {
                l_mub = -l_fb*l_mua/l_fa;
            } else if (l_mean != 0.0f) {
                l_mub = (l_fb - l_mean)*l_mua/(l_mean - l_fa);
            }
            break;
        case 'n':
            l_mub = l_dx;
            if (l_comp == 1) {
                l_mua = -l_fa*l_mub/l_fb;
            } else if (l_mean != 0.0f) {
                l_mua = (l_fa - l_mean)*l_mub/(l_mean - l_fb);
            }
            break;
    }

    //step size & number of steps
    float l_dt;
    int i, l_spp, l_samples;

    l_spp = d_spp;

    if (l_lambda != 0.0f) {
        l_dt = 1.0f/l_lambda/l_spp;
    }

    if (l_mua != 0.0f) {
        float taua, taub;

        taua = 1.0f/l_mua;
        taub = 1.0f/l_mub;

        if (taua < taub) {
            l_dt = taua/l_spp;
        } else {
            l_dt = taub/l_spp;
        }
    }

    l_samples = d_samples;

    //jump countdowns
    int l_pcd, l_dcd, l_dst;
    
    l_pcd = d_pcd[idx];
    l_dcd = d_dcd[idx];
    l_dst = d_dst[idx];

    for (i = 0; i < l_samples; i++) {
        predcorr(l_x, l_x, l_pcd, l_pcd, &l_state, l_Dg, l_Dp, l_lambda, l_comp, \
                 l_dcd, l_dcd, l_dst, l_dst, l_fa, l_fb, l_mua, l_mub, l_dt); 
    }

    //write back path parameters to the global memory
    d_x[idx] = l_x;
    d_pcd[idx] = l_pcd;
    d_dcd[idx] = l_dcd;
    d_dst[idx] = l_dst;
    d_states[idx] = l_state;
}

void prepare()
//prepare simulation
{
    //grid size
    h_paths = (h_paths/h_block)*h_block;
    h_threads = h_paths;

    if (h_moments) h_threads *= h_points;

    h_grid = h_threads/h_block;

    //number of steps
    if (h_moments) h_steps = h_periods*h_spp;
     
    //host memory allocation
    size_f = h_threads*sizeof(float);
    size_i = h_threads*sizeof(int);
    size_ui = h_threads*sizeof(unsigned int);
    size_p = h_points*sizeof(float);

    h_x = (float*)malloc(size_f);
    h_seeds = (unsigned int*)malloc(size_ui);

    //create & initialize host rng
    hiprandCreateGeneratorHost(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));

    hiprandGenerate(gen, h_seeds, h_threads);
 
    //device memory allocation
    hipMalloc((void**)&d_x, size_f);
    hipMalloc((void**)&d_seeds, size_ui);
    hipMalloc((void**)&d_pcd, size_i);
    hipMalloc((void**)&d_dcd, size_i);
    hipMalloc((void**)&d_dst, size_i);
    hipMalloc((void**)&d_states, h_threads*sizeof(hiprandState));

    //copy seeds from host to device
    hipMemcpy(d_seeds, h_seeds, size_ui, hipMemcpyHostToDevice);

    //initialization of device rng
    init_dev_rng<<<h_grid, h_block>>>(d_seeds, d_states);

    free(h_seeds);
    hipFree(d_seeds);

    //moments specific requirements
    if (h_moments) {
        h_trigger = h_steps*h_trans;

        h_xb = (float*)malloc(size_f);
        h_fx = (float*)malloc(size_f);
        h_dx = (float*)malloc(size_p);

        float dxtmp = h_beginx;
        float dxstep = (h_endx - h_beginx)/h_points;

        int i;
        
        //set domainx
        for (i = 0; i < h_points; i++) {
            if (h_logx) {
                h_dx[i] = exp10f(dxtmp);
            } else {
                h_dx[i] = dxtmp;
            }
            dxtmp += dxstep;
        }
        
        hipMalloc((void**)&d_fx, size_f);
        hipMalloc((void**)&d_dx, size_p);
    
        hipMemcpy(d_fx, h_fx, size_f, hipMemcpyHostToDevice);
        hipMemcpy(d_dx, h_dx, size_p, hipMemcpyHostToDevice);
    }
}

void copy_to_dev()
{
    hipMemcpy(d_x, h_x, size_f, hipMemcpyHostToDevice);
}

void copy_from_dev()
{
    hipMemcpy(h_x, d_x, size_f, hipMemcpyDeviceToHost);
}

void initial_conditions()
//set initial conditions for path parameters
{
    int i;

    hiprandGenerateUniform(gen, h_x, h_threads); //x in (0,1]

    for (i = 0; i < h_threads; i++) {
        h_x[i] = 2.0f*h_x[i] - 1.0f; //x in (-1,1]
    }

    copy_to_dev();
}

void moments(float *av)
//calculate the first moment of v
{
    float sx, sxb, tmp, taua, taub, dt;
    int i, j;

    hipMemcpy(h_x, d_x, size_f, hipMemcpyDeviceToHost);
    hipMemcpy(h_fx, d_fx, size_f, hipMemcpyDeviceToHost);

    unfold(h_x, h_fx);

    for (j = 0; j < h_points; j++) {
        sx = 0.0f;
        sxb = 0.0f;

        for (i = 0; i < h_paths; i++) {
            sx += h_x[j*h_paths + i];
            sxb += h_xb[j*h_paths + i];
        }

        //Poissonian
        if (h_domainx == 'l') {
            dt = 1.0f/h_dx[j]/h_spp;
        } else if (h_domainx == 'p' && h_mean != 0.0f) {
            dt = 1.0f/(h_mean*h_mean/h_dx[j])/h_spp;
        } else if (h_lambda != 0.0f) {
            dt = 1.0f/h_lambda/h_spp;
        }

        //Dichotomous
        if (h_domainx == 'm') {
            taua = 1.0f/h_dx[j];
            taub = 1.0f/h_mub;

            if (h_comp) {
                tmp = 1.0f/(-h_fb*h_dx[j]/h_fa);
            } else if (h_mean != 0.0f) {
                tmp = (h_fb - h_mean)*h_dx[j]/(h_mean - h_fa);
            } else {
                tmp = taub;
            }

            if (taua <= tmp) {
                dt = taua/h_spp;
            } else {
                dt = tmp/h_spp;
            }
        } else if (h_domainx == 'n') {
            taua = 1.0f/h_mua;
            taub = 1.0f/h_dx[j];

            if (h_comp) {
                tmp = 1.0f/(-h_fa*h_dx[j]/h_fb);
            } else if (h_mean != 0.0f) {
                tmp = (h_fa - h_mean)*h_dx[j]/(h_mean - h_fb);
            } else {
                tmp = taua;
            }

            if (taub <= tmp) {
                dt = taub/h_spp;
            } else {
                dt = tmp/h_spp;
            }
        } else if (h_mua != 0.0f || h_mub != 0.0f) {
            taua = 1.0f/h_mua;
            taub = 1.0f/h_mub;

            if (taua < taub) {
                dt = taua/h_spp;
            } else {
                dt = taub/h_spp;
            }
        }
            
        av[j] = (sx - sxb)/( (1.0f - h_trans)*h_steps*dt )/h_paths;
    }
}

void finish()
//free memory
{

    free(h_x);
    
    hiprandDestroyGenerator(gen);
    hipFree(d_x);
    hipFree(d_pcd);
    hipFree(d_dcd);
    hipFree(d_dst);
    hipFree(d_states);
    
    if (h_moments) {
        free(h_xb);
        free(h_fx);
        free(h_dx);

        hipFree(d_fx);
        hipFree(d_dx);
    }
}

int main(int argc, char **argv)
{
    parse_cla(argc, argv);
    if (!h_moments) {
        usage(argv);
        return -1;
    }
 
    prepare();
    
    initial_conditions();
    
    //asymptotic long time average velocity <<v>>
    if (h_moments) {
        float *av;
        int i;

        av = (float*)malloc(size_p);
 
        if ( !strcmp(h_domain, "1d") ) { 

            init_noise<<<h_grid, h_block>>>(d_dx, d_pcd, d_dcd, d_dst, d_states);

            for (i = 0; i < h_steps; i += h_samples) {
                run_moments<<<h_grid, h_block>>>(d_x, d_dx, d_pcd, d_dcd, d_dst, d_states);
                fold<<<h_grid, h_block>>>(d_x, d_fx);
                if (i == h_trigger) {
                    hipMemcpy(h_xb, d_x, size_f, hipMemcpyDeviceToHost);
                    hipMemcpy(h_fx, d_fx, size_f, hipMemcpyDeviceToHost);
                    unfold(h_xb, h_fx);
                }
            }

            moments(av);
 
            printf("#%c <<v>>\n", h_domainx);
            for (i = 0; i < h_points; i++) {
                printf("%e %e\n", h_dx[i], av[i]);
            }
        }

        free(av);
    }

    finish();
    
    return 0;
}
