/*
 * Overdamped Brownian particle in symmetric piecewise linear potential
 *
 * \dot{x} = -V'(x) + Gaussian, Poissonian and dichotomous noise
 *
 */

#include <stdio.h>
#include <getopt.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define PI 3.14159265358979f

//model
__constant__ float d_Dg, d_Dp, d_lambda, d_mean, d_fa, d_fb, d_mua, d_mub;
__constant__ int d_comp;
float h_lambda, h_fa, h_fb, h_mua, h_mub, h_mean;
int h_comp;

//simulation
float h_trans;
int h_dev, h_block, h_grid, h_spp;
long h_paths, h_periods, h_threads, h_steps, h_trigger;
__constant__ int d_spp;
__constant__ long d_paths, d_steps, d_trigger;

//output
char *h_domain;
char h_domainx;
float h_beginx, h_endx;
int h_logx, h_points, h_moments;
__constant__ char d_domainx;
__constant__ int d_points;

//vector
float *h_x, *h_xb, *h_dx;
float *d_x, *d_xb, *d_dx;
unsigned int *h_seeds, *d_seeds;
hiprandState *d_states;

size_t size_f, size_ui, size_p;
hiprandGenerator_t gen;

static struct option options[] = {
    {"Dg", required_argument, NULL, 'a'},
    {"Dp", required_argument, NULL, 'b'},
    {"lambda", required_argument, NULL, 'c'},
    {"fa", required_argument, NULL, 'd'},
    {"fb", required_argument, NULL, 'e'},
    {"mua", required_argument, NULL, 'f'},
    {"mub", required_argument, NULL, 'g'},
    {"comp", required_argument, NULL, 'h'},
    {"mean", required_argument, NULL, 'i'},
    {"dev", required_argument, NULL, 'j'},
    {"block", required_argument, NULL, 'k'},
    {"paths", required_argument, NULL, 'l'},
    {"periods", required_argument, NULL, 'm'},
    {"trans", required_argument, NULL, 'n'},
    {"spp", required_argument, NULL, 'o'},
    {"mode", required_argument, NULL, 'p'},
    {"domain", required_argument, NULL, 'q'},
    {"domainx", required_argument, NULL, 'r'},
    {"logx", required_argument, NULL, 's'},
    {"points", required_argument, NULL, 't'},
    {"beginx", required_argument, NULL, 'u'},
    {"endx", required_argument, NULL, 'v'}
};

void usage(char **argv)
{
    printf("Usage: %s <params> \n\n", argv[0]);
    printf("Model params:\n");
    printf("    -a, --Dg=FLOAT          set the Gaussian noise intensity 'D_G' to FLOAT\n");
    printf("    -b, --Dp=FLOAT          set the Poissonian noise intensity 'D_P' to FLOAT\n");
    printf("    -c, --lambda=FLOAT      set the Poissonian kicks frequency '\\lambda' to FLOAT\n");
    printf("    -d, --fa=FLOAT          set the first state of the dichotomous noise 'F_a' to FLOAT\n");
    printf("    -e, --fb=FLOAT          set the second state of the dichotomous noise 'F_b' to FLOAT\n");
    printf("    -f, --mua=FLOAT         set the transition rate of the first state of dichotomous noise '\\mu_a' to FLOAT\n");
    printf("    -g, --mub=FLOAT         set the transition rate of the second state of dichotomous noise '\\mu_b' to FLOAT\n");
    printf("    -h, --comp=INT          choose between biased and unbiased Poissonian or dichotomous noise. INT can be one of:\n");
    printf("                            0: biased; 1: unbiased\n");
    printf("    -i, --mean=FLOAT        if is nonzero, fix the mean value of Poissonian noise or dichotomous noise to FLOAT, matters only for domains p, l, a, b, m or n\n");
    printf("Simulation params:\n");
    printf("    -j, --dev=INT           set the gpu device to INT\n");
    printf("    -k, --block=INT         set the gpu block size to INT\n");
    printf("    -l, --paths=LONG        set the number of paths to LONG\n");
    printf("    -m, --periods=LONG      set the number of periods to LONG\n");
    printf("    -n, --trans=FLOAT       specify fraction FLOAT of periods which stands for transients\n");
    printf("    -o, --spp=INT           specify how many integration steps should be calculated for a single period of the driving force\n");
    printf("Output params:\n");
    printf("    -p, --mode=STRING       sets the output mode. STRING can be one of:\n");
    printf("                            moments: the first moment <<v>>\n");
    printf("    -q, --domain=STRING     simultaneously scan over one or two model params. STRING can be one of:\n");
    printf("                            1d: only one parameter\n");
    printf("    -r, --domainx=CHAR      sets the first domain of the moments. CHAR can be one of:\n");
    printf("                            D: Dg; p: Dp; l: lambda; a: fa; b: fb; m: mua; n: mub\n");
    printf("    -s, --logx=INT          choose between linear and logarithmic scale of the domainx\n");
    printf("                            0: linear; 1: logarithmic\n");
    printf("    -t, --points=INT        set the number of samples to generate between begin and end\n");
    printf("    -u, --beginx=FLOAT      set the starting value of the domainx to FLOAT\n");
    printf("    -v, --endx=FLOAT        set the end value of the domainx to FLOAT\n");
    printf("\n");
}

void parse_cla(int argc, char **argv)
{
    float ftmp;
    int c, itmp;

    while( (c = getopt_long(argc, argv, "a:b:c:d:e:f:g:h:i:j:k:l:m:n:o:p:q:r:s:t:u:v", options, NULL)) != EOF) {
        switch (c) {
            case 'a':
                ftmp = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_Dg), &ftmp, sizeof(float));
                break;
            case 'b':
                ftmp = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_Dp), &ftmp, sizeof(float));
                break;
            case 'c':
                h_lambda = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_lambda), &h_lambda, sizeof(float));
                break;
            case 'd':
                h_fa = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_fa), &h_fa, sizeof(float));
                break;
            case 'e':
                h_fb = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_fb), &h_fb, sizeof(float));
                break;
            case 'f':
                h_mua = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_mua), &h_mua, sizeof(float));
                break;
            case 'g':
                h_mub = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_mub), &h_mub, sizeof(float));
                break;
            case 'h':
                h_comp = atoi(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_comp), &h_comp, sizeof(int));
                break;
            case 'i':
                h_mean = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_mean), &h_mean, sizeof(float));
                break;
            case 'j':
                itmp = atoi(optarg);
                hipSetDevice(itmp);
                break;
            case 'k':
                h_block = atoi(optarg);
                break;
            case 'l':
                h_paths = atol(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_paths), &h_paths, sizeof(long));
                break;
            case 'm':
                h_periods = atol(optarg);
                break;
            case 'n':
                h_trans = atof(optarg);
                break;
            case 'o':
                h_spp = atoi(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_spp), &h_spp, sizeof(int));
                break;
            case 'p':
                if ( !strcmp(optarg, "moments") ) {
                    h_moments = 1;
                }
                break;
            case 'q':
                h_domain = optarg;
                break;
            case 'r':
                h_domainx = optarg[0]; 
                hipMemcpyToSymbol(HIP_SYMBOL(d_domainx), &h_domainx, sizeof(char));
                break;
            case 's':
                h_logx = atoi(optarg);
                break;
            case 't':
                h_points = atoi(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_points), &h_points, sizeof(int));
                break;
            case 'u':
                h_beginx = atof(optarg);
                break;
            case 'v':
                h_endx = atof(optarg);
                break;
            }
    }
}

__global__ void init_dev_rng(unsigned int *d_seeds, hiprandState *d_states)
{
    long idx = blockIdx.x * blockDim.x + threadIdx.x;

    hiprand_init(d_seeds[idx], 0, 0, &d_states[idx]);
}

__device__ float drift(float l_x)
{
    float l_y, l_f;

    l_y = fmod(l_x, 2.0f);

    if (l_y < -1.0f) {
        l_y += 2.0f;
    } else if (l_y > 1.0f) {
        l_y -= 2.0f;
    }

    if (l_y >= -1.0f && l_y < 0.0f) {
        l_f = 1.0f;
    } else if (l_y >= 0.0f && l_y <= 1.0f) {
        l_f = -1.0f;
    }

    return l_f;
}

__device__ float diffusion(float l_Dg, float l_dt, hiprandState *l_state)
{
    if (l_Dg != 0.0f) {
        float r = hiprand_uniform(l_state);
        if ( r <= 1.0f/6 ) {
            return -sqrtf(6.0f*l_Dg*l_dt);
        } else if ( r > 1.0f/6 && r <= 2.0f/6 ) {
            return sqrtf(6.0f*l_Dg*l_dt);
        } else {
            return 0.0f;
        }
    } else {
        return 0.0f;
    }
}

__device__ float adapted_jump_poisson(int &npcd, int pcd, float l_lambda, float l_Dp, int l_comp, float l_dt, hiprandState *l_state)
{
    if (l_lambda != 0.0f) {
        if (pcd <= 0) {
            float ampmean = sqrtf(l_lambda/l_Dp);
           
            npcd = (int) floor( -logf( hiprand_uniform(l_state) )/l_lambda/l_dt + 0.5f );

            if (l_comp) {
                float comp = sqrtf(l_Dp*l_lambda)*l_dt;
                
                return -logf( hiprand_uniform(l_state) )/ampmean - comp;
            } else {
                return -logf( hiprand_uniform(l_state) )/ampmean;
            }
        } else {
            npcd = pcd - 1;
            if (l_comp) {
                float comp = sqrtf(l_Dp*l_lambda)*l_dt;
                
                return -comp;
            } else {
                return 0.0f;
            }
        }
    } else {
        return 0.0f;
    }
}

__device__ float adapted_jump_dich(int &ndcd, int dcd, int &ndst, int dst, float l_fa, float l_fb, float l_mua, float l_mub, float l_dt, hiprandState *l_state)
{
    if (l_mua != 0.0f || l_mub != 0.0f) {
        if (dcd <= 0) {
            if (dst == 0) {
                ndst = 1; 
                ndcd = (int) floor( -logf( hiprand_uniform(l_state) )/l_mub/l_dt + 0.5f );
                return l_fb*l_dt;
            } else {
                ndst = 0;
                ndcd = (int) floor( -logf( hiprand_uniform(l_state) )/l_mua/l_dt + 0.5f );
                return l_fa*l_dt;
            }
        } else {
            ndcd = dcd - 1;
            if (dst == 0) {
                return l_fa*l_dt;
            } else {
                return l_fb*l_dt;
            }
        }
    } else {
        return 0.0f;
    }
}

__device__ void predcorr(float &corrl_x, float l_x, int &npcd, int pcd, hiprandState *l_state, \
                         float l_Dg, float l_Dp, float l_lambda, int l_comp, \
                         int &ndcd, int dcd, int &ndst, int dst, float l_fa, float l_fb, float l_mua, float l_mub, float l_dt)
/* simplified weak order 2.0 adapted predictor-corrector scheme
( see E. Platen, N. Bruti-Liberati; Numerical Solution of Stochastic Differential Equations with Jumps in Finance; Springer 2010; p. 503, p. 532 )
*/
{
    float l_xt, l_xtt, predl_x;

    l_xt = drift(l_x);

    predl_x = l_x + l_xt*l_dt + diffusion(l_Dg, l_dt, l_state);

    l_xtt = drift(predl_x);

    predl_x = l_x + 0.5f*(l_xt + l_xtt)*l_dt + diffusion(l_Dg, l_dt, l_state);

    l_xtt = drift(predl_x);

    corrl_x = l_x + 0.5f*(l_xt + l_xtt)*l_dt + adapted_jump_dich(ndcd, dcd, ndst, dst, l_fa, l_fb, l_mua, l_mub, l_dt, l_state) + diffusion(l_Dg, l_dt, l_state) + adapted_jump_poisson(npcd, pcd, l_lambda, l_Dp, l_comp, l_dt, l_state);
}

__device__ void fold(float &nx, float x, float y, float &nfc, float fc)
//reduce periodic variable to the base domain
{
    float mod;

    mod = floor(x/y)*y;
    nx = x - mod;
    nfc = fc + mod; 
}

__global__ void run_moments(float *d_x, float *d_xb, float *d_dx, hiprandState *d_states)
//actual moments kernel
{
    long idx = blockIdx.x * blockDim.x + threadIdx.x;
    float l_x, l_xb, l_dx; 
    hiprandState l_state;

    //cache path and model parameters in local variables
    l_x = d_x[idx];
    l_xb = d_xb[idx];
    l_state = d_states[idx];

    float l_Dg, l_Dp, l_lambda, l_mean, l_fa, l_fb, l_mua, l_mub;
    int l_comp;

    l_Dg = d_Dg;
    l_Dp = d_Dp;
    l_lambda = d_lambda;
    l_comp = d_comp;
    l_mean = d_mean;
    l_fa = d_fa;
    l_fb = d_fb;
    l_mua = d_mua;
    l_mub = d_mub;

    //run simulation for multiple values of the system parameters
    long ridx = (idx/d_paths) % d_points;
    l_dx = d_dx[ridx];

    switch(d_domainx) {
        case 'D':
            l_Dg = l_dx;
            break;
        case 'p':
            l_Dp = l_dx;
            if (l_mean != 0.0f) l_lambda = (l_mean*l_mean)/l_Dp;
            break;
        case 'l':
            l_lambda = l_dx;
            if (l_mean != 0.0f) l_Dp = (l_mean*l_mean)/l_lambda;
            break;
        case 'a':
            l_fa = l_dx;
            if (l_comp == 1) {
                l_fb = -l_fa*l_mub/l_mua;
            } else if (l_mean != 0.0f) {
                l_fb = (l_mean*(l_mua + l_mub) - l_fa*l_mub)/l_mua;
            }
            break;
        case 'b':
            l_fb = l_dx;
            if (l_comp == 1) {
                l_fa = -l_fb*l_mua/l_mub;
            } else if (l_mean != 0.0f) {
                l_fa = (l_mean*(l_mua + l_mub) - l_fb*l_mua)/l_mub;
            }
            break;
        case 'm':
            l_mua = l_dx;
            if (l_comp == 1) {
                l_mub = -l_fb*l_mua/l_fa;
            } else if (l_mean != 0.0f) {
                l_mub = (l_fb - l_mean)*l_mua/(l_mean - l_fa);
            }
            break;
        case 'n':
            l_mub = l_dx;
            if (l_comp == 1) {
                l_mua = -l_fa*l_mub/l_fb;
            } else if (l_mean != 0.0f) {
                l_mua = (l_fa - l_mean)*l_mub/(l_mean - l_fb);
            }
            break;
    }

    //step size & number of steps
    float l_dt;
    long l_steps, l_trigger, i;

    if (l_lambda != 0.0f) {
        l_dt = 1.0f/l_lambda/d_spp;
    }

    if (l_mua != 0.0f || l_mub != 0.0f) {
        float taua, taub;

        taua = 1.0f/l_mua;
        taub = 1.0f/l_mub;

        if (taua < taub) {
            l_dt = taua/d_spp;
        } else {
            l_dt = taub/d_spp;
        }
    }

    l_steps = d_steps;
    l_trigger = d_trigger;

    //counters for folding
    float xfc;
    
    xfc = 0.0f;

    int pcd, dcd, dst;

    //jump countdowns
    if (l_lambda != 0.0f) pcd = (int) floor( -logf( hiprand_uniform(&l_state) )/l_lambda/l_dt + 0.5f );

    if (l_mua != 0.0f || l_mub != 0.0f) {
        float rn;
        rn = hiprand_uniform(&l_state);

        if (rn < 0.5f) {
            dst = 0;
            dcd = (int) floor( -logf( hiprand_uniform(&l_state) )/l_mua/l_dt + 0.5f);
        } else {
            dst = 1;
            dcd = (int) floor( -logf( hiprand_uniform(&l_state) )/l_mub/l_dt + 0.5f);
        }
    }
    
    for (i = 0; i < l_steps; i++) {

        predcorr(l_x, l_x, pcd, pcd, &l_state, l_Dg, l_Dp, l_lambda, l_comp, \
                 dcd, dcd, dst, dst, l_fa, l_fb, l_mua, l_mub, l_dt);
        
        //fold path parameters
        if ( fabs(l_x) > 2.0f ) {
            fold(l_x, l_x, 2.0f, xfc, xfc);
        }

        if (i == l_trigger) {
            l_xb = l_x + xfc;
        }

    }

    //write back path parameters to the global memory
    d_x[idx] = l_x + xfc;
    d_xb[idx] = l_xb;
    d_states[idx] = l_state;
}

void prepare()
//prepare simulation
{
    //grid size
    h_paths = (h_paths/h_block)*h_block;
    h_threads = h_paths;

    if (h_moments) h_threads *= h_points;

    h_grid = h_threads/h_block;

    //number of steps
    if (h_moments) h_steps = h_periods*h_spp;
    hipMemcpyToSymbol(HIP_SYMBOL(d_steps), &h_steps, sizeof(long));
     
    //host memory allocation
    size_f = h_threads*sizeof(float);
    size_ui = h_threads*sizeof(unsigned int);
    size_p = h_points*sizeof(float);

    h_x = (float*)malloc(size_f);
    h_seeds = (unsigned int*)malloc(size_ui);

    //create & initialize host rng
    hiprandCreateGeneratorHost(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));

    hiprandGenerate(gen, h_seeds, h_threads);
 
    //device memory allocation
    hipMalloc((void**)&d_x, size_f);
    hipMalloc((void**)&d_seeds, size_ui);
    hipMalloc((void**)&d_states, h_threads*sizeof(hiprandState));

    //copy seeds from host to device
    hipMemcpy(d_seeds, h_seeds, size_ui, hipMemcpyHostToDevice);

    //initialization of device rng
    init_dev_rng<<<h_grid, h_block>>>(d_seeds, d_states);

    free(h_seeds);
    hipFree(d_seeds);

    //moments specific requirements
    if (h_moments) {
        h_trigger = h_steps*h_trans;
        hipMemcpyToSymbol(HIP_SYMBOL(d_trigger), &h_trigger, sizeof(long));

        h_xb = (float*)malloc(size_f);
        h_dx = (float*)malloc(size_p);

        float dxtmp = h_beginx;
        float dxstep = (h_endx - h_beginx)/h_points;

        long i;
        
        //set domainx
        for (i = 0; i < h_points; i++) {
            if (h_logx) {
                h_dx[i] = pow(10.0f, dxtmp);
            } else {
                h_dx[i] = dxtmp;
            }
            dxtmp += dxstep;
        }
        
        hipMalloc((void**)&d_xb, size_f);
        hipMalloc((void**)&d_dx, size_p);
    
        hipMemcpy(d_dx, h_dx, size_p, hipMemcpyHostToDevice);
    }
}

void copy_to_dev()
{
    hipMemcpy(d_x, h_x, size_f, hipMemcpyHostToDevice);
    if (h_moments) {
        hipMemcpy(d_xb, h_xb, size_f, hipMemcpyHostToDevice);
    }
}

void copy_from_dev()
{
    hipMemcpy(h_x, d_x, size_f, hipMemcpyDeviceToHost);
    if (h_moments) {
        hipMemcpy(h_xb, d_xb, size_f, hipMemcpyDeviceToHost);
    }
}

void initial_conditions()
//set initial conditions for path parameters
{
    int i;

    hiprandGenerateUniform(gen, h_x, h_threads); //x in (0,1]

    for (i = 0; i < h_threads; i++) {
        h_x[i] = 2.0f*h_x[i] - 1.0f; //x in (-1,1]
    }

    if (h_moments) {
        memset(h_xb, 0, size_f);
    }
    
    copy_to_dev();
}

void moments(float *av)
//calculate the first moment of v
{
    float sx, sxb, tmp, taua, taub, dt;
    int i, j;

    hipMemcpy(h_x, d_x, size_f, hipMemcpyDeviceToHost);
    hipMemcpy(h_xb, d_xb, size_f, hipMemcpyDeviceToHost);

    for (j = 0; j < h_points; j++) {
        sx = 0.0f;
        sxb = 0.0f;

        for (i = 0; i < h_paths; i++) {
            sx += h_x[j*h_paths + i];
            sxb += h_xb[j*h_paths + i];
        }

        //Poissonian
        if (h_domainx == 'l') {
            dt = 1.0f/h_dx[j]/h_spp;
        } else if (h_domainx == 'p' && h_mean != 0.0f) {
            dt = 1.0f/(h_mean*h_mean/h_dx[j])/h_spp;
        } else if (h_lambda != 0.0f) {
            dt = 1.0f/h_lambda/h_spp;
        }

        //Dichotomous
        if (h_domainx == 'm') {
            taua = 1.0f/h_dx[j];
            taub = 1.0f/h_mub;

            if (h_comp) {
                tmp = 1.0f/(-h_fb*h_dx[j]/h_fa);
            } else if (h_mean != 0.0f) {
                tmp = (h_fb - h_mean)*h_dx[j]/(h_mean - h_fa);
            } else {
                tmp = taub;
            }

            if (taua <= tmp) {
                dt = taua/h_spp;
            } else {
                dt = tmp/h_spp;
            }
        } else if (h_domainx == 'n') {
            taua = 1.0f/h_mua;
            taub = 1.0f/h_dx[j];

            if (h_comp) {
                tmp = 1.0f/(-h_fa*h_dx[j]/h_fb);
            } else if (h_mean != 0.0f) {
                tmp = (h_fa - h_mean)*h_dx[j]/(h_mean - h_fb);
            } else {
                tmp = taua;
            }

            if (taub <= tmp) {
                dt = taub/h_spp;
            } else {
                dt = tmp/h_spp;
            }
        } else if (h_mua != 0.0f || h_mub != 0.0f) {
            taua = 1.0f/h_mua;
            taub = 1.0f/h_mub;

            if (taua < taub) {
                dt = taua/h_spp;
            } else {
                dt = taub/h_spp;
            }
        }
            
        av[j] = (sx - sxb)/( (1.0f - h_trans)*h_steps*dt )/h_paths;
    }
}

void finish()
//free memory
{

    free(h_x);
    
    hiprandDestroyGenerator(gen);
    hipFree(d_x);
    hipFree(d_states);
    
    if (h_moments) {
        free(h_xb);
        free(h_dx);

        hipFree(d_xb);
        hipFree(d_dx);
    }
}

int main(int argc, char **argv)
{
    parse_cla(argc, argv);
    if (!h_moments) {
        usage(argv);
        return -1;
    }

    prepare();
    
    initial_conditions();
    
    //asymptotic long time average velocity <<v>>
    if (h_moments) {
        float *av;
        int i;

        av = (float*)malloc(size_p);

        if ( !strcmp(h_domain, "1d") ) {
            run_moments<<<h_grid, h_block>>>(d_x, d_xb, d_dx, d_states);
            moments(av);

            printf("#%c <<v>>\n", h_domainx);
            for (i = 0; i < h_points; i++) {
                printf("%e %e\n", h_dx[i], av[i]);
            }   
        }

        free(av);
    }

    finish();

    return 0;
}
